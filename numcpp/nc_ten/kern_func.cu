#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kern_func.cuh"

template <typename T>
__global__ void add_kernel(const T* a, const T* b, T* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = a[i] + b[i];
    }
}

template <typename T>
void launch_add_kernel(const T* a, const T* b, T* out, int n) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    add_kernel<<<blocks, threads>>>(a, b, out, n);
}
//? I dont know why I need unsigned long  
// Explicit instantiations (important if you only want float/double etc.)
template void launch_add_kernel<int>(const int*, const int*, int*, int);
template void launch_add_kernel<long>(const long*, const long*, long*, int);
template void launch_add_kernel<unsigned long>(const unsigned long*, const unsigned long*, unsigned long*, int);
template void launch_add_kernel<float>(const float*, const float*, float*, int);
template void launch_add_kernel<double>(const double*, const double*, double*, int);
