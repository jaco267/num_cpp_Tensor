#include "hip/hip_runtime.h"
// func.cu
#include <hip/hip_runtime.h>
#include "func.cuh"
#include <iostream>
using std::cout; 
using std::endl;
// CUDA kernel
__global__ void addKernel(const int* a, const int* b, int* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

// wrapper implementation
extern "C" void addArrays(const int* a, const int* b, int* c, int n) {
    int *d_a, *d_b, *d_c;
    // size_t size = n * sizeof(int);
    cout<<"add array....!!!"<<endl;
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
